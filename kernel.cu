#include <cstdio>
#include <hip/hip_runtime.h>

// GPU를 위한 커널 프로그램(NVCC가 컴파일함)
__global__ void addKernel(int* c, const int * a, const int * b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__host__ int main(void){
	const int SIZE = 5;
	const int a[SIZE] = { 1,2,3,4,5 };
	const int b[SIZE] = { 10,20,30,40,50 };
	int c[SIZE] = { 0 };

	//디바이스(GPU) 측 데이터
	int * dev_a = 0;
	int * dev_b = 0;
	int * dev_c = 0;

	//VRAM에 메모리 할당
	hipMalloc((void**)&dev_a, SIZE * sizeof(int));
	hipMalloc((void**)&dev_b, SIZE * sizeof(int));
	hipMalloc((void**)&dev_c, SIZE * sizeof(int));

	//값 복사(host -> device)
	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	//커널 함수 수행
	//dev_c = dev_a + dev_b
	addKernel <<<1, SIZE >>> (dev_c, dev_a, dev_b);

	//값 복사(device -> host)
	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	//메모리 공간 해제
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	//결과 출력
	printf("{%d, %d, %d, %d, %d} + {%d, %d, %d, %d, %d} = {%d, %d, %d, %d, %d}\n",
		a[0], a[1], a[2], a[3], a[4],
		b[0], b[1], b[2], b[3], b[4],
		c[0], c[1], c[2], c[3], c[4]);

	return 0;
}