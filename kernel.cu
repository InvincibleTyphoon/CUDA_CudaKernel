#include <cstdio>
#include <hip/hip_runtime.h>

// GPU�� ���� Ŀ�� ���α׷�(NVCC�� ��������)
__global__ void addKernel(int* c, const int * a, const int * b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__host__ int main(void){
	const int SIZE = 5;
	const int a[SIZE] = { 1,2,3,4,5 };
	const int b[SIZE] = { 10,20,30,40,50 };
	int c[SIZE] = { 0 };

	//����̽�(GPU) �� ������
	int * dev_a = 0;
	int * dev_b = 0;
	int * dev_c = 0;

	//VRAM�� �޸� �Ҵ�
	hipMalloc((void**)&dev_a, SIZE * sizeof(int));
	hipMalloc((void**)&dev_b, SIZE * sizeof(int));
	hipMalloc((void**)&dev_c, SIZE * sizeof(int));

	//�� ����(host -> device)
	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	//Ŀ�� �Լ� ����
	//dev_c = dev_a + dev_b
	addKernel <<<1, SIZE >>> (dev_c, dev_a, dev_b);

	//�� ����(device -> host)
	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	//�޸� ���� ����
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	//��� ���
	printf("{%d, %d, %d, %d, %d} + {%d, %d, %d, %d, %d} = {%d, %d, %d, %d, %d}\n",
		a[0], a[1], a[2], a[3], a[4],
		b[0], b[1], b[2], b[3], b[4],
		c[0], c[1], c[2], c[3], c[4]);

	return 0;
}